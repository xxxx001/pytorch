#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/Dispatch_v2.h>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/_local_scalar_dense_native.h>
#endif

#include <ATen/cuda/HIPContext.h>

namespace at::native {

Scalar _local_scalar_dense_cuda(const Tensor& self) {
  Scalar r;
  AT_DISPATCH_V2(
    self.scalar_type(), "_local_scalar_dense_cuda", AT_WRAP([&] {
        scalar_t value;
        hipStream_t stream = at::cuda::getCurrentCUDAStream();
        at::cuda::memcpy_and_sync(&value, self.const_data_ptr<scalar_t>(), sizeof(scalar_t), hipMemcpyDeviceToHost, stream);
        r = Scalar(value);
      }), AT_EXPAND(AT_ALL_TYPES_AND_COMPLEX), kComplexHalf, kHalf, kBool, kBFloat16, AT_EXPAND(AT_BAREBONES_UNSIGNED_TYPES));
  return r;
}

} // at::native
